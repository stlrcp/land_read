#include "hip/hip_runtime.h"


#ifdef __ILUVATAR__
#define POS_INFINITY INFINITY
#define NEG_INFINITY -INFINITY
#else
#define POS_INFINITY __int_as_float(0x7f800000)
#define INFINITY POS_INFINITY
#define NEG_INFINITY __int_as_float(0xff800000)
#define NAN __int_as_float(0x7fffffff)
#endif


#ifndef __ILUVATAR__
// typedef long long int int64_t;
typedef unsigned long long int uint64_t;
#endif



#define NVFUSER_UPDATE_MAGIC_ZERO \
  do {                            \
    nvfuser_zero <<= 1;           \
  } while (0);

#define NVFUSER_DEFINE_MAGIC_ZERO          \
  __shared__ int nvfuser_zero_s;           \
  if (threadIdx.x == 0)                    \
    nvfuser_zero_s = 0;                    \
  __syncthreads();                         \
  atomicMin(&nvfuser_zero_s, threadIdx.x); \
  int nvfuser_zero = nvfuser_zero_s;


__device__ constexpr int ceilDiv(int a, int b) {
  return (a + b - 1) / b;
}

__device__ constexpr int64_t ceilDiv(int64_t a, int64_t b) {
  return (a + b - 1) / b;
}

__device__ constexpr int64_t ceilDiv(int64_t a, int b) {
  return ceilDiv(a, (int64_t)b);
}

__device__ constexpr int64_t ceilDiv(int a, int64_t b) {
  return ceilDiv((int64_t)a, b);
}

#ifndef __ILUVATAR__
__device__ constexpr double ceilDiv(double a, double b) {
  return std::ceil(a / b);
}
#else
__device__ constexpr float ceilDiv(float a, float b) {
  return std::ceil(a / b);
}
#endif

#ifndef __ILUVATAR__
__device__ constexpr double ceilDiv(double a, int64_t b) {
  return std::ceil(a / b);
}
#else
__device__ constexpr float ceilDiv(float a, int64_t b) {
  return std::ceil(a / b);
}
#endif

#ifndef __ILUVATAR__
__device__ constexpr double ceilDiv(int64_t a, double b) {
  return std::ceil(a / b);
}
#else
__device__ constexpr float ceilDiv(int64_t a, float b) {
  return std::ceil(a / b);
}
#endif


template <typename T, int N>
struct Tensor {
  __device__ T& operator[](int ind) {
    return data[ind];
  };

  T* data;
  int size[N];
  int stride[N];
};

// Specialization for 0-dim case as it does not need size and stride arrays.
// They will be an error as well since zero-length arrays are not allowed.
template <typename T>
struct Tensor<T, 0> {
  __device__ T& operator[](int) {
    return *data;
  };

  T* data;
};


namespace block_sync {

__forceinline__ __device__ void init() {}

// Thread-block synchronization
__forceinline__ __device__ void sync() {
  __syncthreads();
}

} // namespace block_sync


namespace index_utils {

// Utility functions

// Total size of provided dimension
template <typename _dim3>
__device__ __forceinline__ int size(const _dim3& d) {
  return (int)d.x * (int)d.y * (int)d.z;
}

// Linearized indexing of idx based on dim, if bool==false that dimension does
// not participate
template <bool X, bool Y, bool Z, typename _dim3, typename _dim3_2>
__device__ int maskedOffset(const _dim3& idx, const _dim3_2& dim) {
  int offset = 0;
  if (Z)
    offset += idx.z;
  if (Y)
    offset = offset * dim.y + idx.y;
  if (X)
    offset = offset * dim.x + idx.x;
  return offset;
}

// Linearized indexing of idx based on dim. All dimensions participate.
template <typename _dim3, typename _dim3_2>
__device__ int offset(const _dim3& idx, const _dim3_2& dim) {
  int offset = idx.z;
  offset = offset * dim.y + idx.y;
  offset = offset * dim.x + idx.x;
  return offset;
}

// Masks the provided dim3, those == false get truncated to 1
template <bool X, bool Y, bool Z, typename _dim3>
__device__ dim3 maskedDims(const _dim3& dim) {
  return dim3{
      X ? (unsigned)dim.x : 1U,
      Y ? (unsigned)dim.y : 1U,
      Z ? (unsigned)dim.z : 1U};
}

// Provides total size of dim with masking, those dims == false do not
// participate in the size calculation
template <bool X_BLOCK, bool Y_BLOCK, bool Z_BLOCK, typename _dim3>
__device__ int maskedSize(const _dim3& dim) {
  return size(maskedDims<X_BLOCK, Y_BLOCK, Z_BLOCK>(dim));
}

// Checks if provided idx is zero on those dims == true
template <bool X, bool Y, bool Z, typename _dim3>
__device__ bool maskedIsZero(const _dim3& idx) {
  bool isZero = true;
  if (X)
    isZero = isZero && idx.x == 0;
  if (Y)
    isZero = isZero && idx.y == 0;
  if (Z)
    isZero = isZero && idx.z == 0;
  return isZero;
}

// Checks if provided idx is zero on those dims == true
template <bool X, bool Y, bool Z, typename _dim3, typename _dim3_2>
__device__ bool maskedIsLast(const _dim3& idx, const _dim3_2& dim) {
  bool isZero = true;
  if (X)
    isZero = isZero && idx.x == dim.x - 1;
  if (Y)
    isZero = isZero && idx.y == dim.y - 1;
  if (Z)
    isZero = isZero && idx.z == dim.z - 1;
  return isZero;
}

} // namespace index_utils


namespace grid_sync {

// Get the first bit in a 64 bit integer
#define FIRST_UINT64_BIT ((uint64_t)1 << (sizeof(uint64_t) * 8 - 1))
#define LOW_UINT64_BIT ((uint32_t)((FIRST_UINT64_BIT) & 0xFFFFFFFF))
#define HIGH_UINT64_BIT ((uint32_t)(((FIRST_UINT64_BIT) >> 32) & 0xFFFFFFFF))

template <typename T>
__device__ T globalAsVolatile(volatile T& global_val) {
  return global_val;
}

// A grid synchronization that can be called multiple times in a kernel assuming
// all the blocks fit on device at once. The semaphore is an integer semaphore
// assumed to be initialized to 0 before launching the kernel. The persistent
// option should be envoked if this sync will be called multiple times in one
// kernel (i.e. having a grid reduce within a loop). Having multiple grid syncs
// called once in the same kernel does not require persistent mode. Segment size
// is the number of blocks participating in the sync in the dimensions marked by
// [X,Y,Z]_BLOCK. The granularity of this sync are those dimensions. I.E.
// Marking X and Y but not Z means there should be Z semaphores of size X*Y.
template <bool X_BLOCK, bool Y_BLOCK, bool Z_BLOCK, bool PERSISTENT>
__device__ void sync(
    int64_t& semaphore,
    const uint64_t& segment_size,
    const bool last_block) {
  // Finish all global memory transactions before synchronizing
  __threadfence();

  // Synchronize all threads in a block before synchronizing blocks
  block_sync::sync();

  // Only allow linear_tid == 0 to participate in the synchronization
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    // Get increment value, only want a single block to have the large
    // increment, doesn't really matter which one, the goal is to flip/flop the
    // first bit of a uint64_t value, since our semaphores are actualy int64_t
    // we will just reinterpret_cast it to act as a uint64_t
    uint64_t semaphore_increment = 1;

    // Makes the assumption that blocks are in increasing order, this is not
    // guaranteed by CUDA but this is the current behavior, and unlikely to
    // change.
    if (last_block) {
      semaphore_increment = FIRST_UINT64_BIT - (segment_size - 1);
    }

  #ifdef __ILUVATAR__
    // operate last_block after other blocks complete atomicAdd() (will fail if segment_size > 0x7FFFFFFF)
    uint2 *address_uint2 = reinterpret_cast<uint2 *>(&semaphore);
    const uint2 &val_uint2 = reinterpret_cast<uint2 &>(semaphore_increment);
    uint2 old_val_uint2;
    if (!last_block) {
        old_val_uint2.x = atomicAdd(&((*address_uint2).x), val_uint2.x);
        if (old_val_uint2.x + val_uint2.x < val_uint2.x) {
            old_val_uint2.y = atomicAdd(&((*address_uint2).y), (1 + val_uint2.y));
        } else {
            old_val_uint2.y = atomicAdd(&((*address_uint2).y), val_uint2.y);
        }
    } else {
        while (globalAsVolatile(semaphore) != (segment_size - 1)) {
        }
        old_val_uint2.x = atomicExch(&((*address_uint2).x), LOW_UINT64_BIT);
        old_val_uint2.y = atomicExch(&((*address_uint2).y), HIGH_UINT64_BIT);
    }
    uint64_t oldArrive = reinterpret_cast<uint64_t &>(old_val_uint2);
  #else
    uint64_t oldArrive =
        atomicAdd(reinterpret_cast<uint64_t*>(&semaphore), semaphore_increment);
  #endif

    // If for persistent kernels, lock all blocks until the semaphore has been
    // reached. Make sure we access semaphore as a volatile address so we get
    // the global memory updates.
    unsigned int ns = 8;
    while ((PERSISTENT || last_block) &&
           ((oldArrive ^ globalAsVolatile(semaphore)) & FIRST_UINT64_BIT) ==
               0) {
      // Put a sleep here so we have some breaks in probing the global
      // semaphore, giving a better chance for other warps/blocks to catch up.
#if __CUDA_ARCH__ >= 700
      // __nanosleep only available on compute capability 7.0 or higher
      __nanosleep(ns); // avoids busy waiting
      if (ns < 256) {
        ns *= 2;
      }
#endif
    }
  }

  // Sync block to make sure all other threads are waiting on the sync
  block_sync::sync();
}

template <bool X_BLOCK, bool Y_BLOCK, bool Z_BLOCK, bool PERSISTENT>
__device__ void sync(int64_t& semaphore, const uint64_t& segment_size) {
  sync<X_BLOCK, Y_BLOCK, Z_BLOCK, PERSISTENT>(
      semaphore,
      segment_size,
      index_utils::maskedIsLast<X_BLOCK, Y_BLOCK, Z_BLOCK>(blockIdx, gridDim));
}

// Grid sync that can be called multiple times in the same kernel without all
// blocks being resident on device. This allows grid sync to be called multiple
// times as long as it's not broadcasted on the parallel axis it was reduced on.
//
// n_entrances is how many times every block is expected to enter into this
// function. All blocks must enter n_entrances times. The last block is only
// allowed to proceed once all other blocks have entered n_entrance
// times.
//
// Note that this is not currently used by grid and welford reduction
// as they use a separate sync flag for each each grid sync call.
template <bool X_BLOCK, bool Y_BLOCK, bool Z_BLOCK>
__device__ void sync(
    int64_t& semaphore,
    const uint64_t& segment_size,
    const int n_entrances) {
  // Finish all global memory transactions before synchronizing
  __threadfence();

  // Synchronize all threads in a block before synchronizing blocks
  block_sync::sync();

  // Only allow linear_tid == 0 to participate in the synchronization
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    // Makes the assumption that blocks are in increasing order, this is not
    // guaranteed by CUDA but this is the current behavior, and unlikely to
    // change.
    bool last_block =
        index_utils::maskedIsLast<X_BLOCK, Y_BLOCK, Z_BLOCK>(blockIdx, gridDim);
    if (last_block) {
      int64_t finished_val =
          ((int64_t)(
              index_utils::maskedSize<X_BLOCK, Y_BLOCK, Z_BLOCK>(gridDim) -
              1)) *
          ((int64_t)n_entrances);

      unsigned int ns = 8;
      // Last block needs to wait for all other blocks to finish
      while (globalAsVolatile(semaphore) < finished_val) {
#if __CUDA_ARCH__ >= 700
        // __nanosleep only available on compute capability 7.0 or higher
        __nanosleep(ns); // avoids busy waiting
        if (ns < 256) {
          ns *= 2;
        }
#endif
      }
    } else {
#ifdef __ILUVATAR__
      auto old = atomicAdd(reinterpret_cast<uint32_t*>(&semaphore), 1);
#else
      auto old = atomicAdd(reinterpret_cast<uint64_t*>(&semaphore), 1);
#endif
    }
  }

  // Sync block to make sure all other threads are waiting on the sync
  block_sync::sync();
}

} // namespace grid_sync




template <bool X_REDUCE, bool Y_REDUCE, bool Z_REDUCE, typename T, typename Func, typename _dim3, typename _dim3_2>
__device__ void blockReduce(T &out, const T &inp_val, Func reduction_op, const _dim3 &thread_idx,
    const _dim3_2 &block_dim, T *shared_mem, bool read_pred, bool write_pred, T init_val)
{
    // If this thread will output a final result
    bool should_write =
        index_utils::maskedIsZero<X_REDUCE, Y_REDUCE, Z_REDUCE>(thread_idx);

    // Size of the reduction segments
    unsigned int reduction_size =
        index_utils::maskedSize<X_REDUCE, Y_REDUCE, Z_REDUCE>(block_dim);

    // Index into the reduction segment
    unsigned int reduction_tid =
        index_utils::maskedOffset<X_REDUCE, Y_REDUCE, Z_REDUCE>(
            thread_idx, block_dim);

    // Index of the reduction segment
    unsigned int reduction_idx =
        index_utils::maskedOffset<!X_REDUCE, !Y_REDUCE, !Z_REDUCE>(
            thread_idx, block_dim);

    // Offset into smem for the current thread
    unsigned int smem_offset = reduction_idx * reduction_size + reduction_tid;

    // Initialize shared memory
    if (read_pred)
    {
        shared_mem[smem_offset] = inp_val;
    }
    else
    {
        shared_mem[smem_offset] = init_val;
    }

    block_sync::sync();
    // Reduce down to nearest power of 2 for the tree reduction:
    int np2 = 1 << (31 - __clz(reduction_size));

    if (reduction_tid < np2 && reduction_tid + np2 < reduction_size)
    {
        reduction_op(shared_mem[smem_offset], shared_mem[smem_offset + np2]);
    }
    block_sync::sync();

    // loop peel the final iteration to save one syncthread for the end
    for (int factor = np2 / 2; factor > 1; factor >>= 1)
    {
        if (reduction_tid < factor)
        {
            reduction_op(shared_mem[smem_offset], shared_mem[smem_offset + factor]);
        }
        block_sync::sync();
    }

    if (should_write && write_pred)
    {
        T result = out;
        reduction_op(result, shared_mem[smem_offset]);
        if (reduction_size > 1)
        {
            reduction_op(result, shared_mem[smem_offset + 1]);
        }
        out = result;
    }
    block_sync::sync();
}

// Use the same pred for both reads and writes
template <bool X_REDUCE, bool Y_REDUCE, bool Z_REDUCE, typename T, typename Func, typename _dim3, typename _dim3_2>
__device__ void blockReduce(T &out, const T &inp_val, Func reduction_op, const _dim3 &thread_idx, 
    const _dim3_2 &block_dim, T *shared_mem, bool read_write_pred, T init_val)
{
    blockReduce<X_REDUCE, Y_REDUCE, Z_REDUCE, T, Func, _dim3, _dim3_2>(
        out,
        inp_val,
        reduction_op,
        thread_idx,
        block_dim,
        shared_mem,
        read_write_pred,
        read_write_pred,
        init_val);
}


namespace reduction {

// Reduces all the reduction blocks in each reduction segment. This is the
// "cleanup" stage of a grid reduction.
//
// This is only called by one thread block per reduction segment. The input
// reduction blocks of the segment are stored in an intermediate buffer pointed
// by parameter in. Template parameters X/Y/Z_THREAD denote how the reduction
// block is formed.
//
// The size of a reduction block is by definition smaller or equal to the size
// of a thread block. We use the remaining threads to parallelize reductions
// across reduction blocks. For example, when X/Y/Z_THREAD = {true, false,
// false}, we use blockDim.y*blockDim.z threads for each output value. This is
// done first by loading the input values in parallel and then by reducing
// across threads of dimensions whose XYZ_THREAD are false.
//
// Note that what is done here after the loading from global memory is similar
// to what the existing blockReduce function does.
template <
    bool X_THREAD,
    bool Y_THREAD,
    bool Z_THREAD,
    typename T,
    typename Func>
__device__ void gridReduceLastBlock(
    T& out,
    const volatile T* in,
    const int
        grid_reduction_segment_size, // Number of reductions across
                                     // grid reduce dimensions
    const int
        block_reduction_segment_size, // Number of reductions across the block
    Func reduction_op,
    T* shared_buf,
    bool write_pred,
    T init_val) {
  // We have to do num_reductions across reduction_size. The reductions are
  // contiguous, but offset by reduction_size. There is an entry in "in" for
  // every block, and every thread marked as true. Threads in dimensions marked
  // as false can be used to parallelize the reduction.

  // Find the reduction id of the participating threads
  const auto block_reduction_segment_idx =
      index_utils::maskedOffset<X_THREAD, Y_THREAD, Z_THREAD>(
          threadIdx, blockDim);

  // Find an id associated within a reduction segment for all
  // "non-participating" threads, which will parallelize the reductions for the
  // "participating" threads
  const auto id_in_block_segment =
      index_utils::maskedOffset<!X_THREAD, !Y_THREAD, !Z_THREAD>(
          threadIdx, blockDim);

  // Stride by the "non-participating" threads
  const auto input_stride_for_thread_in_segment =
      index_utils::maskedSize<!X_THREAD, !Y_THREAD, !Z_THREAD>(blockDim);

  T inp = init_val;

  // Block stride across the reduction until we only have one value per thread
  for (int reduction_i = id_in_block_segment;
       reduction_i < grid_reduction_segment_size;
       reduction_i += input_stride_for_thread_in_segment) {
    auto work_buf_offset = reduction_i * block_reduction_segment_size +
        block_reduction_segment_idx;
    reduction_op(inp, in[work_buf_offset]);
  }

  // Block reduce the per thread values into per "participating" thread values
  T inp_tmp = init_val;
  blockReduce<!X_THREAD, !Y_THREAD, !Z_THREAD>(
      inp_tmp,
      inp,
      reduction_op,
      threadIdx,
      blockDim,
      shared_buf,
      true,
      init_val);
  const bool should_write = (X_THREAD || threadIdx.x == 0) &&
      (Y_THREAD || threadIdx.y == 0) && (Z_THREAD || threadIdx.z == 0);
  if (should_write && write_pred) {
    reduction_op(out, inp_tmp);
  }
}

// Reduces per-thread values across threads and thread blocks.
//
// Function parameters:
// - out: Per-thread output location
// - inp_val: Per-thread input value
// - reduction_op: Scalar reduction function
// - work_buf: Temporary buffer for cross-block reductions
// - sync_flags: A vector of integers for synchronizations
// - shared_buf: Shared memory buffer for intra-block reduction
//
// Thread has valid results based on if it's the last block in the grid
// reduction dimension
//
// Template parameters:
// - X/Y/Z_BLOCK/THREAD: When true, reduces across thread blocks along the X/Y/Z
//   dimensions
// - PERSISTENT_REDUCTION: Indicates grid reduction will be called in a loop, or
//   the result of the grid reduction will be broadcasted and used across the
//   grid. These requires cross grid communication and the grid synchronizations
//   here to actually synchronize across the entire grid. When false the grid is
//   not synchronized, the last block just waits for everyone else to finish and
//   the other blocks can exit early.
// - T: Scalar data type of input/output data
// - Func: Type of scalara reduction function
//
// Template parameters X/Y/Z_BLOCK define a group of thread blocks that are
// reduced together. We call it a reduction segment. Some examples are:
//
// Case 1: X/Y/Z_BLOCK == true/true/true -> There is only one segment, which
// includes all thread blocks. It is effecively the same as the grid.
//
// Case 2: X/Y/Z_BLOCK == false/false/false -> Each thread block comprises an
// individual segment by itself.
//
// Case 3: X/Y/Z_BLOCK == true/false/false -> Each segment contains thread
// blocks that have the same blockDim.x. There will be blockDim.y*blockDim.z
// such segments.
//
// X/Y/Z_THREAD also works similarly as X/Y/Z_BLOCK and defines a
// group of threads that are reduced togather.
//
// After the function completes, only one thread block per reduction segment
// gets valid reduction results. There is no guarantee which particular block
// gets the final results.
//
// entrance_ind and n_entrances are allowed when PERSISTENT_REDUCTION = false.
// If a grid reduction call is only called once per thread, entrance_ind == 0
// and n_entrances == 1. However, grid reduction can be called in a loop in a
// thread, in that case entrance_ind is the count of times the function has been
// called, and n_entrances is the total number of times it will be called.
template <
    bool X_BLOCK,
    bool Y_BLOCK,
    bool Z_BLOCK,
    bool X_THREAD,
    bool Y_THREAD,
    bool Z_THREAD,
    bool PERSISTENT_REDUCTION,
    typename T,
    typename Func>
__device__ void gridReduce(
    T& out,
    const T& inp_val,
    Func reduction_op,
    volatile T* work_buf,
    int64_t* sync_flags,
    T* shared_buf,
    bool read_pred,
    bool write_pred,
    T init_val,
    const int entrance_ind,
    const int n_entrances) {
  T block_reduction_val = init_val;

  // Do block reduction when required
  if (X_THREAD || Y_THREAD || Z_THREAD) {
    blockReduce<X_THREAD, Y_THREAD, Z_THREAD>(
        block_reduction_val,
        inp_val,
        reduction_op,
        threadIdx,
        blockDim,
        shared_buf,
        read_pred,
        true,
        init_val);
  } else if (read_pred) {
    block_reduction_val = inp_val;
  }

  // Number of values to reduce in the reduction segment
  const auto grid_reduction_segment_size =
      index_utils::maskedSize<X_BLOCK, Y_BLOCK, Z_BLOCK>(gridDim);

  // Index of the reduction we're performing out of the
  // grid_reduction_segment_size
  const auto idx_in_grid_segment =
      index_utils::maskedOffset<!X_BLOCK, !Y_BLOCK, !Z_BLOCK>(
          blockIdx, gridDim);

  // Number of threads we can use in final reduction, Seems to assume all
  // threads in the block participate
  const auto block_reduction_segment_size =
      index_utils::maskedSize<!X_THREAD, !Y_THREAD, !Z_THREAD>(blockDim);

  // Number of reductions in the grid
  const int grid_segment_size = PERSISTENT_REDUCTION
      ? 1
      : index_utils::maskedSize<!X_BLOCK, !Y_BLOCK, !Z_BLOCK>(gridDim);

  // advance to the offset for this segment
  // index of reduction * size of the reduction * size of threads
  work_buf += (entrance_ind * grid_segment_size + idx_in_grid_segment) *
      grid_reduction_segment_size * block_reduction_segment_size;

  if ((!X_THREAD || threadIdx.x == 0) && (!Y_THREAD || threadIdx.y == 0) &&
      (!Z_THREAD || threadIdx.z == 0)) {
    auto block_offset =
        index_utils::maskedOffset<X_BLOCK, Y_BLOCK, Z_BLOCK>(blockIdx, gridDim);
    auto thread_offset =
        index_utils::maskedOffset<!X_THREAD, !Y_THREAD, !Z_THREAD>(
            threadIdx, blockDim);
    auto work_buf_offset =
        block_offset * block_reduction_segment_size + thread_offset;
    work_buf[work_buf_offset] = block_reduction_val;
  }
  if (PERSISTENT_REDUCTION) {
    grid_sync::sync<X_BLOCK, Y_BLOCK, Z_BLOCK, PERSISTENT_REDUCTION>(
        sync_flags[idx_in_grid_segment], grid_reduction_segment_size);

  } else {
    // Use a different sync flag for each call
    grid_sync::sync<X_BLOCK, Y_BLOCK, Z_BLOCK, PERSISTENT_REDUCTION>(
        sync_flags[entrance_ind * grid_segment_size + idx_in_grid_segment],
        grid_reduction_segment_size);
  }

  bool last_block =
      index_utils::maskedIsLast<X_BLOCK, Y_BLOCK, Z_BLOCK>(blockIdx, gridDim);

  if (last_block) {
    // Cleanup with block reduction
    gridReduceLastBlock<!X_THREAD, !Y_THREAD, !Z_THREAD>(
        out,
        (T*)work_buf,
        grid_reduction_segment_size,
        block_reduction_segment_size,
        reduction_op,
        shared_buf,
        write_pred,
        init_val);
  }

  if (PERSISTENT_REDUCTION) {
    // Make sure we're done with global memory before we allow the kernel to
    // continue
    grid_sync::sync<X_BLOCK, Y_BLOCK, Z_BLOCK, PERSISTENT_REDUCTION>(
        sync_flags[idx_in_grid_segment], grid_reduction_segment_size);
  }
}

// This is just a wrapper of the above grid reduction routine to
// measure the elapsed cycles. The measurement must be done just by
// one thread, and in this case it should be done by one of the
// threads in the last thread block.
#ifdef PYTORCH_NVFUSER_PROFILE_KERNEL
template <
    bool X_BLOCK,
    bool Y_BLOCK,
    bool Z_BLOCK,
    bool X_THREAD,
    bool Y_THREAD,
    bool Z_THREAD,
    bool PERSISTENT_REDUCTION,
    typename T,
    typename Func>
__device__ void gridReduce(
    T& out,
    const T& inp_val,
    Func reduction_op,
    volatile T* work_buf,
    int64_t* sync_flags,
    T* shared_buf,
    bool read_pred,
    bool write_pred,
    T init_val,
    const int entrance_ind,
    const int n_entrances,
    int64_t& cycles,
    int64_t& count) {
  int64_t start_counter = 0;

  if (index_utils::maskedIsLast<true, true, true>(blockIdx, gridDim) &&
      index_utils::maskedIsZero<true, true, true>(threadIdx)) {
    start_counter = readCycleCounter();
  }

  gridReduce<
      X_BLOCK,
      Y_BLOCK,
      Z_BLOCK,
      X_THREAD,
      Y_THREAD,
      Z_THREAD,
      PERSISTENT_REDUCTION,
      T,
      Func>(
      out,
      inp_val,
      reduction_op,
      work_buf,
      sync_flags,
      shared_buf,
      read_pred,
      write_pred,
      init_val,
      entrance_ind,
      n_entrances);

  if (index_utils::maskedIsLast<true, true, true>(blockIdx, gridDim) &&
      index_utils::maskedIsZero<true, true, true>(threadIdx)) {
    cycles += readCycleCounter() - start_counter;
    ++count;
  }
}
#endif // PYTORCH_NVFUSER_PROFILE_KERNEL

template <
    bool X_BLOCK,
    bool Y_BLOCK,
    bool Z_BLOCK,
    bool X_THREAD,
    bool Y_THREAD,
    bool Z_THREAD,
    typename T,
    typename Func>
__device__ void gridReduce2PartialReduction(
    const T& inp_val,
    T init_val,
    Func reduction_op,
    volatile T* work_buf,
    T* shared_buf,
    bool read_pred,
    int grid_reduction_segment_size,
    int idx_in_grid_segment,
    int block_reduction_segment_size) {
  T block_reduction_val = init_val;

  // Do block reduction when required
  if (X_THREAD || Y_THREAD || Z_THREAD) {
    blockReduce<X_THREAD, Y_THREAD, Z_THREAD>(
        block_reduction_val,
        inp_val,
        reduction_op,
        threadIdx,
        blockDim,
        shared_buf,
        read_pred,
        true,
        init_val);
  } else if (read_pred) {
    block_reduction_val = inp_val;
  }

  if ((!X_THREAD || threadIdx.x == 0) && (!Y_THREAD || threadIdx.y == 0) &&
      (!Z_THREAD || threadIdx.z == 0)) {
    auto block_offset =
        index_utils::maskedOffset<X_BLOCK, Y_BLOCK, Z_BLOCK>(blockIdx, gridDim);
    auto thread_offset =
        index_utils::maskedOffset<!X_THREAD, !Y_THREAD, !Z_THREAD>(
            threadIdx, blockDim);
    auto work_buf_offset =
        block_offset * block_reduction_segment_size + thread_offset;
    work_buf[work_buf_offset] = block_reduction_val;
  }
}

// 2-way horizontally fused grid reduction
template <
    bool X_BLOCK,
    bool Y_BLOCK,
    bool Z_BLOCK,
    bool X_THREAD,
    bool Y_THREAD,
    bool Z_THREAD,
    bool PERSISTENT_REDUCTION,
    typename T1,
    typename Func1,
    typename T2,
    typename Func2>
__device__ void gridReduceGroup(
    T1& out1,
    const T1& inp_val1,
    T1 init_val1,
    Func1 reduction_op1,
    volatile T1* work_buf1,
    T2& out2,
    const T2& inp_val2,
    T2 init_val2,
    Func2 reduction_op2,
    volatile T2* work_buf2,
    int64_t* sync_flags,
    void* shared_buf,
    bool read_pred,

    bool write_pred,
    const int entrance_ind,
    const int n_entrances) {
  // Number of values to reduce in the reduction segment
  const auto grid_reduction_segment_size =
      index_utils::maskedSize<X_BLOCK, Y_BLOCK, Z_BLOCK>(gridDim);

  // Index of the reduction we're performing out of the
  // grid_reduction_segment_size
  const auto idx_in_grid_segment =
      index_utils::maskedOffset<!X_BLOCK, !Y_BLOCK, !Z_BLOCK>(
          blockIdx, gridDim);

  // Number of threads we can use in final reduction, Seems to assume all
  // threads in the block participate
  const auto block_reduction_segment_size =
      index_utils::maskedSize<!X_THREAD, !Y_THREAD, !Z_THREAD>(blockDim);

  // Number of reductions in the grid
  const int grid_segment_size = PERSISTENT_REDUCTION
      ? 1
      : index_utils::maskedSize<!X_BLOCK, !Y_BLOCK, !Z_BLOCK>(gridDim);

  // advance to the offset for this segment
  // index of reduction * size of the reduction * size of threads
  work_buf1 += (entrance_ind * grid_segment_size + idx_in_grid_segment) *
      grid_reduction_segment_size * block_reduction_segment_size;

  work_buf2 += (entrance_ind * grid_segment_size + idx_in_grid_segment) *
      grid_reduction_segment_size * block_reduction_segment_size;

  gridReduce2PartialReduction<
      X_BLOCK,
      Y_BLOCK,
      Z_BLOCK,
      X_THREAD,
      Y_THREAD,
      Z_THREAD>(
      inp_val1,
      init_val1,
      reduction_op1,
      work_buf1,
      (T1*)shared_buf,
      read_pred,
      grid_reduction_segment_size,
      idx_in_grid_segment,
      block_reduction_segment_size);

  gridReduce2PartialReduction<
      X_BLOCK,
      Y_BLOCK,
      Z_BLOCK,
      X_THREAD,
      Y_THREAD,
      Z_THREAD>(
      inp_val2,
      init_val2,
      reduction_op2,
      work_buf2,
      (T2*)shared_buf,
      read_pred,
      grid_reduction_segment_size,
      idx_in_grid_segment,
      block_reduction_segment_size);

  if (PERSISTENT_REDUCTION) {
    grid_sync::sync<X_BLOCK, Y_BLOCK, Z_BLOCK, PERSISTENT_REDUCTION>(
        sync_flags[idx_in_grid_segment], grid_reduction_segment_size);
  } else {
    grid_sync::sync<X_BLOCK, Y_BLOCK, Z_BLOCK, PERSISTENT_REDUCTION>(
        sync_flags[entrance_ind * grid_segment_size + idx_in_grid_segment],
        grid_reduction_segment_size);
  }

  bool last_block =
      index_utils::maskedIsLast<X_BLOCK, Y_BLOCK, Z_BLOCK>(blockIdx, gridDim);

  if (last_block) {
    // Cleanup with block reduction
    gridReduceLastBlock<!X_THREAD, !Y_THREAD, !Z_THREAD>(
        out1,
        work_buf1,
        grid_reduction_segment_size,
        block_reduction_segment_size,
        reduction_op1,
        (T1*)shared_buf,
        write_pred,
        init_val1);
    gridReduceLastBlock<!X_THREAD, !Y_THREAD, !Z_THREAD>(
        out2,
        work_buf2,
        grid_reduction_segment_size,
        block_reduction_segment_size,
        reduction_op2,
        (T2*)shared_buf,
        write_pred,
        init_val2);
  }

  if (PERSISTENT_REDUCTION) {
    // Make sure we're done with global memory before we allow the kernel to
    // continue
    grid_sync::sync<X_BLOCK, Y_BLOCK, Z_BLOCK, PERSISTENT_REDUCTION>(
        sync_flags[idx_in_grid_segment], grid_reduction_segment_size);
  }
}

#ifdef PYTORCH_NVFUSER_PROFILE_KERNEL
template <
    bool X_BLOCK,
    bool Y_BLOCK,
    bool Z_BLOCK,
    bool X_THREAD,
    bool Y_THREAD,
    bool Z_THREAD,
    bool PERSISTENT_REDUCTION,
    typename T1,
    typename Func1,
    typename T2,
    typename Func2>
__device__ void gridReduceGroup(
    T1& out1,
    const T1& inp_val1,
    T1 init_val1,
    Func1 reduction_op1,
    volatile T1* work_buf1,
    T2& out2,
    const T2& inp_val2,
    T2 init_val2,
    Func2 reduction_op2,
    volatile T2* work_buf2,
    int64_t* sync_flags,
    void* shared_buf,
    bool read_pred,
    bool write_pred,
    const int entrance_ind,
    const int n_entrances,
    int64_t& cycles,
    int64_t& count) {
  int64_t start_counter = 0;

  if (index_utils::maskedIsLast<true, true, true>(blockIdx, gridDim) &&
      index_utils::maskedIsZero<true, true, true>(threadIdx)) {
    start_counter = readCycleCounter();
  }

  gridReduceGroup<
      X_BLOCK,
      Y_BLOCK,
      Z_BLOCK,
      X_THREAD,
      Y_THREAD,
      Z_THREAD,
      PERSISTENT_REDUCTION,
      T1,
      Func1,
      T2,
      Func2>(
      out1,
      inp_val1,
      init_val1,
      reduction_op1,
      work_buf1,
      out2,
      inp_val2,
      init_val2,
      reduction_op2,
      work_buf2,
      sync_flags,
      shared_buf,
      read_pred,
      write_pred,
      entrance_ind,
      n_entrances);

  if (index_utils::maskedIsLast<true, true, true>(blockIdx, gridDim) &&
      index_utils::maskedIsZero<true, true, true>(threadIdx)) {
    cycles += readCycleCounter() - start_counter;
    ++count;
  }
}
#endif // PYTORCH_NVFUSER_PROFILE_KERNEL

} // namespace reduction








__global__ void kernel1(Tensor<float, 3> T0, Tensor<float, 2> T2, Tensor<float, 1> T4, Tensor<int64_t, 1> T5) {
  alignas(16) extern __shared__ char array[];
  void* shared_mem = array;
  // Allocate global tensor T4
  // Allocate global tensor T5
  int i61;
  i61 = (((int)blockIdx.x) * ((int)blockDim.x)) + ((int)threadIdx.x);
  float T3[1];
  T3[0] = NEG_INFINITY;
  #pragma unroll 1
  for(int i43 = 0; i43 < (ceilDiv((ceilDiv((ceilDiv(T0.size[0], ((int)blockDim.y))), 1)), ((int)gridDim.y))); ++i43) {
    int i59;
    i59 = (((((int)blockIdx.y) * (ceilDiv((ceilDiv((ceilDiv(T0.size[0], ((int)blockDim.y))), 1)), ((int)gridDim.y)))) + i43) * ((int)blockDim.y)) + ((int)threadIdx.y);
    if (((i59 < T0.size[0]) && (i61 < (T0.size[1] * T0.size[2])))) {
      float T1[1];
      T1[0] = NEG_INFINITY;
      T1[0]
        = T0[(i59 * (T0.size[2] * T0.size[1])) + i61]
        * (float) 2.00000000000000000e+00;
      T3[0] = fmax(
        T3[0],
        T1[0]);
    }
  }
  if ((i61 < (T0.size[1] * T0.size[2]))) {
    T2[i61] = NEG_INFINITY;
  }
  reduction::gridReduce<false, true, false, false, true, false, false>(
    T2[i61],
    T3[0],
    [](float &a, float b) { a = fmax(a, b); },
    &T4[0],
    &T5[0],
    static_cast<float*>(shared_mem),
    (i61 < (T0.size[1] * T0.size[2])),
    (i61 < (T0.size[1] * T0.size[2])),
    float(NEG_INFINITY),
    0,
    1);
}